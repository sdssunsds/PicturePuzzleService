#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <>
#include <stdio.h>
#include<vector>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/extrema.h>
#include <thrust/functional.h>
#include "write.h"
__global__ void get_float(uint8_t* data, float* float_data, const int h, const int w)
{
	const int n = threadIdx.x;
	const int ID = blockIdx.x * blockDim.x + n;
	if (ID < h)
	{
		//printf("blockIdx.x: %d ,blockDim.x: %d ,n:%d \n", blockIdx.x, blockDim.x, blockIdx.x * blockDim.x + n);
		for (int j = 0; j < w; j++)
		{
			float_data[ID * w + j]=float(data[ID * w + j]) / 255.0;
		//	float num= float(data[ID * w + j]) / 255.0;			
		//	if (num>0.5)
		//	{
		//		float_data[ID * w + j] = 1-tanh(1 - num);
		//	}
		//	else
		//	{
		//		float_data[ID * w + j] = num;
		//	}

		}
	}
}
__global__ void stand_img(float* data, double mean, double var, int h, int w)
{
	const int n = threadIdx.x;
	if (blockIdx.x * blockDim.x + n < h)
	{
		for (int j = 0; j < w; j++)
		{
			int  ID = (blockIdx.x * blockDim.x + n) * w + j;
			data[ID] = (data[ID] - mean) / var;
		}
	}

}
__global__ void get_res(float* data,bool* bool_data ,uint8_t* imgres, double min_num, double max_num, int h, int w)
{
	const int n = threadIdx.x;
	double handel = max_num - min_num;
	if (blockIdx.x * blockDim.x + n < h)
	{
		for (int j = 0; j < w; j++)
		{
			if (blockIdx.x * blockDim.x + n < h)
			{
				float f = (data[(blockIdx.x * blockDim.x + n) * w + j] - min_num) / handel;
				float k = 1 - f<0.4 ? 0.4:1 - f;
				k = bool_data[(blockIdx.x * blockDim.x + n) * w + j] ? k - 0.1 : k;
				imgres[(blockIdx.x * blockDim.x + n) * w + j] = (uchar)(round(tanh(f *5 *k) * 255));
				//imgres[(blockIdx.x * blockDim.x + n) * w + j] = (uchar)(round(tanh((data[(blockIdx.x * blockDim.x + n) * w + j] - min) / handel * 3) * 255));

			}
		}
	}
}
__global__ void get_bool(uchar* data,bool* output,int kernel_size,int h, int w)
{
	const int x =blockIdx.x*blockDim.x+ threadIdx.x;
	const int y = blockIdx.y;
	if (x<w)
	{		
		float l = float(data[x + y * w]);
		float k = float(data[x + y * w]);
		int xmin = x - kernel_size < 0 ? 0 : x - kernel_size;
		int xmax = x + kernel_size < w ? x+kernel_size : w;
		int ymin = y - kernel_size < 0 ? 0 : y - kernel_size;
		int ymax = y + kernel_size < h ? y+kernel_size : h;
		for (int i = xmin; i < xmax; i++)
		{
			for (int j = ymin; j < ymax; j++)
			{
				k = k + data[i + j * w];
			}
		}
		k = k / ((xmax - xmin) * (ymax - ymin));
		output[x + y * w] = bool(k-l >30);
	}
	
}
struct variance : std::unary_function<float, float>
{
	variance(float m) : mean(m) { } const float mean;
	__host__ __device__ float operator()(float data) const
	{
		return ::pow(data - mean, 2.0f);
	}
};

void write_new(cv::Mat& input, cv::Mat& output)
{
	const int h_h = input.rows;
	const int h_w = input.cols;
	const int threadnum = 1024;
	//�����Դ沢��ͼ���ƽ��Դ�
	int blocksize = threadnum;
	int gridsize = (input.rows - 1) / threadnum + 1;
	dim3 grid_size(h_w/ threadnum, h_h);
	int  size_image = input.cols * input.rows;
	int size_image_fp = input.cols * input.rows * sizeof(float);
	uint8_t* data;
	hipMalloc((void**)&data, size_image_fp);
	hipMemcpy(data, input.data, size_image, hipMemcpyHostToDevice);
	float* float_data;
	hipMalloc((void**)&float_data, size_image_fp);
	bool* bool_data;
	hipMalloc((void**)&bool_data, input.cols * input.rows * sizeof(bool));
	get_bool <<<grid_size, blocksize >>> (data, bool_data, 4, h_h, h_w);
	
	get_float <<<gridsize, blocksize >>> (data, float_data, h_h, h_w);
	thrust::device_ptr<float> img_D_vector(float_data);

	double mean = thrust::reduce(img_D_vector, img_D_vector + size_image) / (size_image);
	float var = thrust::transform_reduce(img_D_vector, img_D_vector + size_image, variance(mean), 0.0f, thrust::plus<double>())/ size_image;
	stand_img <<<gridsize, blocksize >>> (float_data, mean, var, h_h, h_w);
	auto max_num = thrust::max_element(img_D_vector, img_D_vector + size_image);
	double max_val = *max_num;
	auto min_num = thrust::min_element(img_D_vector, img_D_vector + size_image);
	double min_val = *min_num;
	//	//�������ͼ���ڴ���Դ�
	uint8_t* res_img_d;
	uint8_t* res_data = (uint8_t*)malloc(size_image);
	hipMalloc((void**)&res_img_d, size_image);
	get_res << <gridsize, blocksize >> > (float_data, bool_data, res_img_d, min_val, max_val, h_h, h_w);
	hipMemcpy(res_data, res_img_d, size_image, hipMemcpyDeviceToHost);
	cv::Mat image(h_h, h_w, CV_8UC1, res_data);
	//image.copyTo(output);
	output = image;
	//��������ݸ�ֵ��ͼ��
	hipFree(data);
	hipFree(bool_data);
	hipFree(float_data);
	hipFree(res_img_d);
}

